#include "hip/hip_runtime.h"
#define DLL
#include "cuda_extension.cuh"
using namespace std;

int SelectedCudaDeviceId;

DLLAPI __global__ bool SetCudaDevice(int device_id)
{
	if (hipSetDevice(device_id) == hipErrorInvalidDevice)
		return false;
	return true;
}

DLLAPI __global__ INIT_CUDA_RETURN_VALUES InitializationCuda()
{
	int cuda_device_count;
	hipGetDeviceCount(&cuda_device_count);
	if (cuda_device_count == 0)
		return NO_DEVICE;
	bool device_select_successfully;
	for (int i = 0; i < cuda_device_count; i++)
	{
		hipDeviceProp_t current_device_properties;
		if (hipGetDeviceProperties(&current_device_properties, i) != hipSuccess)
			return GET_DEVICE_PROPERTIES_FAILED;
		else
			if (current_device_properties.major > 3)
				if (!SetCudaDevice(i))
					return SET_DEVICE_FAILED;
				else
				{
					device_select_successfully = true;
					SelectedCudaDeviceId = i;
					break;
				}
	}
	if (!device_select_successfully)
		return NO_DEVICE_SUPPORTED_LOWEST_VERSION_CUDA;
	return INIT_SUCCESSFULLY;
}

DLLAPI __device__ void CudaAddByte(__global__ char* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ char result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, char);
	*dest = result;
}

DLLAPI __device__ void CudaAddWord(__global__ short* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ short result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, short);
	*dest = result;
}

DLLAPI __device__ void CudaAddDword(__global__ int* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ int result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, int);
	*dest = result;
}

DLLAPI __device__ void CudaAddQword(__global__ long long* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long long result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, long long);
	*dest = result;
}

DLLAPI __device__ void CudaAddFloat(__global__ float* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ float result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, float);
	*dest = result;
}

DLLAPI __device__ void CudaAddDouble(__global__ double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ double result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, double);
	*dest = result;
}

DLLAPI __device__ void CudaAddLongDouble(__global__ long double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long double result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, long double);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractByte(__global__ char* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ char result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, char);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractWord(__global__ short* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ short result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, short);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractDword(__global__ int* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ int result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, int);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractQword(__global__ long long* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long long result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, long long);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractLongDouble(__global__ long double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long double result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, long double);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractDouble(__global__ double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ double result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, double);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractFloat(__global__ float* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ float result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, float);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyByte(__global__ char* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ char result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, char);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyWord(__global__ short* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ short result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, short);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyDword(__global__ int* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ int result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, int);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyQword(__global__ long long* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long long result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, long long);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyFloat(__global__ float* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ float result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, float);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyDouble(__global__ double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ double result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, double);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyLongDouble(__global__ long double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long double result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, long double);
	*dest = result;
}

DLLAPI __device__ void CudaDivideByte(__global__ char* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ char result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, char);
	*dest = result;
}

DLLAPI __device__ void CudaDivideWord(__global__ short* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ short result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, short);
	*dest = result;
}

DLLAPI __device__ void CudaDivideDword(__global__ int* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ int result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, int);
	*dest = result;
}

DLLAPI __device__ void CudaDivideQword(__global__ long long* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long long result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, long long);
	*dest = result;
}

DLLAPI __device__ void CudaDivideFloat(__global__ float* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ float result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, float);
	*dest = result;
}

DLLAPI __device__ void CudaDivideDouble(__global__ double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ double result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, double);
	*dest = result;
}

DLLAPI __device__ void CudaDivideLongDouble(__global__ long double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long double result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, long double);
	*dest = result;
}

DLLAPI __device__ void CudaMatrixByte(__global__ char* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaMatrixWord(__global__ short* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaMatrixDword(__global__ int* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaMatrixQword(__global__ long long* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaMatrixFloat(__global__ float* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaMatrixDouble(__global__ double* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaMatrixLongDouble(__global__ long double* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaCubeByte(__global__ char* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaCubeWord(__global__ short* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaCubeDword(__global__ int* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaCubeQword(__global__ long long* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaCubeFloat(__global__ float* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaCubeDouble(__global__ double* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaCubeLongDouble(__global__ long double* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaPowerByte(__global__ char* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaPowerWord(__global__ short* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaPowerDword(__global__ int* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaPowerQword(__global__ long long* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaPowerFloat(__global__ float* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaPowerDouble(__global__ double* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaPowerLongDouble(__global__ long double* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaSquareRootQword(__global__ long long *dest)
{

}

INT APIENTRY DllMain(HMODULE hModule, DWORD fdwReason, LPVOID lpvReserved)
{
	switch (fdwReason)
	{
	case DLL_PROCESS_ATTACH:
		break;
	case DLL_PROCESS_DETACH:
		break;
	case DLL_THREAD_ATTACH:
		break;
	case DLL_THREAD_DETACH:
		break;
	}
}