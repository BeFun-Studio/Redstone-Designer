#include "hip/hip_runtime.h"
#define DLL
#include "cuda_extension.cuh"
using namespace std;

DLLAPI __global__ bool SetCudaDevice(int device_id)
{
	if (hipSetDevice(device_id) == hipErrorInvalidDevice)
		return false;
	return true;
}

DLLAPI __global__ INIT_CUDA_RETURN_VALUES InitializationCuda()
{
	int cuda_device_count;
	hipGetDeviceCount(&cuda_device_count);
	if (cuda_device_count == 0)
		return NO_DEVICE;
	int j = 0;
	for (int i = 0; i < cuda_device_count; i++)
	{
		hipDeviceProp_t current_device_properties;
		if (hipGetDeviceProperties(&current_device_properties, i) != hipSuccess)
			return GET_DEVICE_PROPERTIES_FAILED;
		else
			if (current_device_properties.major > 3)
				if (!SetCudaDevice(i))
					return SET_DEVICE_FAILED;
	}
	if ( j == 0)
		return NO_DEVICE_SUPPORTED_LOWEST_VERSION_CUDA;
	return INIT_SUCCESSFULLY;
}

DLLAPI __global__ long long CudaAdd(unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	long long result=0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, long long);
	return result;
}

DLLAPI __global__ long long CudaSubtract(unsigned short count, long long original_num, ...)
{
	va_list args;
	va_start(args, count);
	long long result = original_num;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, long long);
	return result;
}

DLLAPI __global__ long long CudaMultiply(unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	long long result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, long long);
	return result;
}

DLLAPI __global__ long long CudaDivide(unsigned short count, long long original_num, ...)
{
	va_list args;
	va_start(args, count);
	long long result = original_num;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, long long);
	return result;
}

INT APIENTRY DllMain(HMODULE hModule, DWORD fdwReason, LPVOID lpvReserved)
{
	switch (fdwReason)
	{
	case DLL_PROCESS_ATTACH:
		break;
	case DLL_PROCESS_DETACH:
		break;
	case DLL_THREAD_ATTACH:
		break;
	case DLL_THREAD_DETACH:
		break;
	}
}