#include "hip/hip_runtime.h"
#define DLL
#include "cuda_extension.cuh"
using namespace std;

int SelectedCudaDeviceId;

DLLAPI __global__ bool SetCudaDevice(int device_id)
{
	if (hipSetDevice(device_id) == hipErrorInvalidDevice)
		return false;
	return true;
}

DLLAPI __global__ INIT_CUDA_RETURN_VALUES InitializationCuda()
{
	int cuda_device_count;
	hipGetDeviceCount(&cuda_device_count);
	if (cuda_device_count == 0)
		return NO_DEVICE;
	bool device_select_successfully;
	for (int i = 0; i < cuda_device_count; i++)
	{
		hipDeviceProp_t current_device_properties;
		if (hipGetDeviceProperties(&current_device_properties, i) != hipSuccess)
			return GET_DEVICE_PROPERTIES_FAILED;
		else
			if (current_device_properties.major > 3)
				if (!SetCudaDevice(i))
					return SET_DEVICE_FAILED;
				else
				{
					device_select_successfully = true;
					SelectedCudaDeviceId = i;
					break;
				}
	}
	if (!device_select_successfully)
		return NO_DEVICE_SUPPORTED_LOWEST_VERSION_CUDA;
	return INIT_SUCCESSFULLY;
}

DLLAPI __device__ void CudaAddByte(__global__ char* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ char result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, char);
	*dest = result;
}

DLLAPI __device__ void CudaAddWord(__global__ short* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ short result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, short);
	*dest = result;
}

DLLAPI __device__ void CudaAddDword(__global__ int* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ int result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, int);
	*dest = result;
}

DLLAPI __device__ void CudaAddQword(__global__ long long* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long long result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, long long);
	*dest = result;
}

DLLAPI __device__ void CudaAddFloat(__global__ float* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ float result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, float);
	*dest = result;
}

DLLAPI __device__ void CudaAddDouble(__global__ double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ double result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, double);
	*dest = result;
}

DLLAPI __device__ void CudaAddLongDouble(__global__ long double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long double result = 0;
	for (unsigned short i = 0; i < count; i++)
		result += va_arg(args, long double);
	*dest = result;
}

DLLAPI __device__ void CudaAddByteList(__global__ char* dest, unsigned short count, __global__ char* lst)
{
	__device__ char result = 0;
	for (__device__ unsigned short i = 0; i < count; i++)
		result += lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaAddWordList(__global__ short* dest, unsigned short count, __global__ short* lst)
{
	__device__ short result = 0;
	for (__device__ short i = 0; i < count; i++)
		result += lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaAddDwordList(__global__ int* dest, unsigned short count, __global__ int* lst)
{
	__device__ int result = 0;
	for (__device__ short i = 0; i < count; i++)
		result += lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaAddQwordList(__global__ long long* dest, unsigned short count, __global__ long long* lst)
{
	__device__ long long result = 0;
	for (__device__ short i = 0; i < count; i++)
		result += lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaAddFloatList(__global__ float* dest, unsigned short count, __global__ float* lst)
{
	__device__ float result = 0;
	for (__device__ short i = 0; i < count; i++)
		result += lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaAddDoubleList(__global__ double* dest, unsigned short count, __global__ double* lst)
{
	__device__ double result = 0;
	for (__device__ short i = 0; i < count; i++)
		result += lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaAddLongDoubleList(__global__ long double* dest, unsigned short count, __global__ long double* lst)
{
	__device__ long double result = 0;
	for (__device__ short i = 0; i < count; i++)
		result += lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaSubtractByte(__global__ char* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ char result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, char);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractWord(__global__ short* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ short result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, short);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractDword(__global__ int* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ int result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, int);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractQword(__global__ long long* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long long result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, long long);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractLongDouble(__global__ long double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long double result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, long double);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractDouble(__global__ double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ double result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, double);
	*dest = result;
}

DLLAPI __device__ void CudaSubtractFloat(__global__ float* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ float result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result -= va_arg(args, float);
	*dest = result;
}


DLLAPI __device__ void CudaSubtractByteList(__global__ char* dest, unsigned short count, __global__ char* lst)
{
	__device__ char result = *dest;
	for (__device__ short i = 0; i < count; i++)
		result -= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaSubtractWordList(__global__ short* dest, unsigned short count, __global__ short* lst)
{
	__device__ short result = *dest;
	for (__device__ short i = 0; i < count; i++)
		result -= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaSubtractDwordList(__global__ int* dest, unsigned short count, __global__ int* lst)
{
	__device__ int result = *dest;
	for (__device__ short i = 0; i < count; i++)
		result -= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaSubtractQwordList(__global__ long long* dest, unsigned short count, __global__ long long* lst)
{
	__device__ long long result = *dest;
	for (__device__ short i = 0; i < count; i++)
		result -= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaSubtractFloatList(__global__ float* dest, unsigned short count, __global__ float* lst)
{
	__device__ float result = *dest;
	for (__device__ short i = 0; i < count; i++)
		result -= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaSubtractDoubleList(__global__ double* dest, unsigned short count, __global__ double* lst)
{
	__device__ double result = *dest;
	for (__device__ short i = 0; i < count; i++)
		result -= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaSubtractLongDoubleList(__global__ long double* dest, unsigned short count, __global__ long double* lst)
{
	__device__ long double result = *dest;
	for (__device__ short i = 0; i < count; i++)
		result -= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyByte(__global__ char* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ char result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, char);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyWord(__global__ short* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ short result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, short);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyDword(__global__ int* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ int result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, int);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyQword(__global__ long long* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long long result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, long long);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyFloat(__global__ float* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ float result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, float);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyDouble(__global__ double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ double result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, double);
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyLongDouble(__global__ long double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long double result = 0;
	for (unsigned short i = 0; i < count; i++)
		result *= va_arg(args, long double);
	*dest = result;
}


DLLAPI __device__ void CudaMultiplyByteList(__global__ char* dest, unsigned short count, __global__ char* lst)
{
	__device__ char result = lst[0];
	for (__device__ unsigned short i = 1; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyWordList(__global__ short* dest, unsigned short count, __global__ short* lst)
{
	__device__ short result = lst[0];
	for (__device__ unsigned short i = 1; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyDwordList(__global__ int* dest, unsigned short count, __global__ int* lst)
{
	__device__ int result = lst[0];
	for (__device__ unsigned short i = 1; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyQwordList(__global__ long long* dest, unsigned short count, __global__ long long* lst)
{
	__device__ long long result = lst[0];
	for (__device__ unsigned short i = 1; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyFloatList(__global__ float* dest, unsigned short count, __global__ float* lst)
{
	__device__ float result = lst[0];
	for (__device__ unsigned short i = 1; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyDoubleList(__global__ double* dest, unsigned short count, __global__ double* lst)
{
	__device__ double result = lst[0];
	for (__device__ unsigned short i = 1; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaMultiplyLongDoubleList(__global__ long double* dest, unsigned short count, __global__ long double* lst)
{
	__device__ long double result = lst[0];
	for (__device__ unsigned short i = 1; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaDivideByte(__global__ char* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ char result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, char);
	*dest = result;
}

DLLAPI __device__ void CudaDivideWord(__global__ short* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ short result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, short);
	*dest = result;
}

DLLAPI __device__ void CudaDivideDword(__global__ int* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ int result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, int);
	*dest = result;
}

DLLAPI __device__ void CudaDivideQword(__global__ long long* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long long result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, long long);
	*dest = result;
}

DLLAPI __device__ void CudaDivideFloat(__global__ float* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ float result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, float);
	*dest = result;
}

DLLAPI __device__ void CudaDivideDouble(__global__ double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ double result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, double);
	*dest = result;
}

DLLAPI __device__ void CudaDivideLongDouble(__global__ long double* dest, unsigned short count, ...)
{
	va_list args;
	va_start(args, count);
	__device__ long double result = *dest;
	for (unsigned short i = 0; i < count; i++)
		result /= va_arg(args, long double);
	*dest = result;
}

DLLAPI __device__ void CudaDivideByteList(__global__ char* dest, unsigned short count, __global__ char* lst)
{
	__device__ char result = *dest;
	for (__device__ unsigned short i = 0; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaDivideWordList(__global__ short* dest, unsigned short count, __global__ short* lst)
{
	__device__ short result = *dest;
	for (__device__ unsigned short i = 0; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaDivideDwordList(__global__ int* dest, unsigned short count, __global__ int* lst)
{
	__device__ int result = *dest;
	for (__device__ unsigned short i = 0; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaDivideQwordList(__global__ long long* dest, unsigned short count, __global__ long long* lst)
{
	__device__ long long result = *dest;
	for (__device__ unsigned short i = 0; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaDivideFloatList(__global__ float* dest, unsigned short count, __global__ float* lst)
{
	__device__ float result = *dest;
	for (__device__ unsigned short i = 0; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaDivideDoubleList(__global__ double* dest, unsigned short count, __global__ double* lst)
{
	__device__ double result = *dest;
	for (__device__ unsigned short i = 0; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaDivideLongDoubleList(__global__ long double* dest, unsigned short count, __global__ long double* lst)
{
	__device__ long double result = *dest;
	for (__device__ unsigned short i = 0; i < count; i++)
		result *= lst[i];
	*dest = result;
}

DLLAPI __device__ void CudaMatrixByte(__global__ char* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaMatrixWord(__global__ short* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaMatrixDword(__global__ int* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaMatrixQword(__global__ long long* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaMatrixFloat(__global__ float* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaMatrixDouble(__global__ double* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaMatrixLongDouble(__global__ long double* dest)
{
	*dest *= *dest;
}

DLLAPI __device__ void CudaCubeByte(__global__ char* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaCubeWord(__global__ short* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaCubeDword(__global__ int* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaCubeQword(__global__ long long* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaCubeFloat(__global__ float* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaCubeDouble(__global__ double* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaCubeLongDouble(__global__ long double* dest)
{
	*dest *= (*dest) * (*dest);
}

DLLAPI __device__ void CudaPowerByte(__global__ char* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaPowerWord(__global__ short* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaPowerDword(__global__ int* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaPowerQword(__global__ long long* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaPowerFloat(__global__ float* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaPowerDouble(__global__ double* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaPowerLongDouble(__global__ long double* dest, unsigned int t)
{
	for (unsigned int i = 0; i < t; i++)
		*dest *= *dest;
}

DLLAPI __device__ void CudaSquareRootQword(__global__ long long *dest)
{

}

INT APIENTRY DllMain(HMODULE hModule, DWORD fdwReason, LPVOID lpvReserved)
{
	switch (fdwReason)
	{
	case DLL_PROCESS_ATTACH:
		break;
	case DLL_PROCESS_DETACH:
		break;
	case DLL_THREAD_ATTACH:
		break;
	case DLL_THREAD_DETACH:
		break;
	}
}